//------------------------------------------------------------------------------
//
// Name:       gameoflife.cu
// 
// Purpose:    CUDA implementation of Conway's game of life
//
// HISTORY:    Written by Tom Deakin and Simon McIntosh-Smith, August 2013
//
//------------------------------------------------------------------------------

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define FINALSTATEFILE "final_state.dat"

// Define the state of the cell
#define DEAD  0
#define ALIVE 1

/*************************************************************************************
 * Forward declarations of utility functions
 ************************************************************************************/
void die(const char* message, const int line, const char *file);
void load_board(char* board, const char* file, const unsigned int nx, const unsigned int ny);
void print_board(const char* board, const unsigned int nx, const unsigned int ny);
void save_board(const char* board, const unsigned int nx, const unsigned int ny);
void load_params(const char *file, unsigned int *nx, unsigned int *ny, unsigned int *iterations);
void errorCheck(hipError_t error);

/*************************************************************************************
 * Game of Life worker method - CUDA kernel
 ************************************************************************************/

// Apply the rules of life to tick and save in tock
__global__ void accelerate_life(const char* tick, char* tock, const int nx, const int ny)
{
    // The cell we work on in the loop
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int idy = blockDim.y * blockIdx.y + threadIdx.y;

    // Copy block to shared memory
    // TODO
    extern __shared__ char block[];

    // Indexes of rows/columns next to idx
    // wrapping around if required
    unsigned int x_l, x_r, y_u, y_d;

    // Calculate indexes
    unsigned int id = idy * nx + idx;
    x_r = (idx + 1) % nx;
    x_l = (idx == 0) ? nx - 1 : idx - 1;
    y_u = (idy + 1) % ny;
    y_d = (idy == 0) ? ny - 1: idy - 1;

    // Count alive neighbours (out of eight)
    int neighbours = 0;
    if (tick[idy * nx + x_l] == ALIVE) neighbours++;
    if (tick[y_u * nx + x_l] == ALIVE) neighbours++;
    if (tick[y_d * nx + x_l] == ALIVE) neighbours++;
        
    if (tick[idy * nx + x_r] == ALIVE) neighbours++;
    if (tick[y_u * nx + x_r] == ALIVE) neighbours++;
    if (tick[y_d * nx + x_r] == ALIVE) neighbours++;
         
    if (tick[y_u * nx + idx] == ALIVE) neighbours++;
    if (tick[y_d * nx + idx] == ALIVE) neighbours++;

    // Apply game of life rules
    if (tick[id] == ALIVE)
    {
        if (neighbours == 2 || neighbours == 3)
            // Cell lives on
            tock[id] = ALIVE;
        else
            // Cell dies by over/under population
            tock[id] = DEAD;
    }
    else
    {
        if (neighbours == 3)
            // Cell becomes alive through reproduction
            tock[id] = ALIVE;
        else
            // Remains dead
            tock[id] = DEAD;
    }

}


/*************************************************************************************
 * Main function
 ************************************************************************************/

int main(int argc, char **argv)
{

    // Check we have a starting state file
    if (argc != 3)
    {
        printf("Usage:\n./gameoflife input.dat input.params\n");
        return EXIT_FAILURE;
    }


    // Board dimensions and iteration total
    unsigned int nx, ny;
    unsigned int iterations;

    load_params(argv[2], &nx, &ny, &iterations);

    // Allocate memory for boards
    size_t size = nx * ny * sizeof(char);
    char* h_board = (char *)calloc(nx * ny, sizeof(char));
    char* d_board_tick;
    char* d_board_tock;

    errorCheck(hipMalloc(&d_board_tick, size));
    errorCheck(hipMalloc(&d_board_tock, size));

    // Load in the starting state to board_tick
    load_board(h_board, argv[1], nx, ny);

    // Display the starting state
    printf("Starting state\n");
    print_board(h_board, nx, ny);

    // Copy the host array to the device array
    errorCheck(hipMemcpy(d_board_tick, h_board, size, hipMemcpyHostToDevice));

    // Define our problem size for CUDA
    dim3 numBlocks(nx/3, ny/3);
    dim3 numThreads(3, 3);
    size_t sharedMem = sizeof(char) * nx * 1;

    // Loop
    for (unsigned int i = 0; i < iterations; i++)
    {
        // Apply the rules of Life
        accelerate_life<<<numBlocks, numThreads, sharedMem>>>(d_board_tick, d_board_tock, nx, ny);

        // Swap the boards over
        char *tmp = d_board_tick;
        d_board_tick = d_board_tock;
        d_board_tock = tmp;
    }

    // Copy the device array back to the host
    errorCheck(hipMemcpy(h_board, d_board_tick, size, hipMemcpyDeviceToHost));

    // Display the final state
    printf("Finishing state\n");
    print_board(h_board, nx, ny);

    // Save the final state of the board
    save_board(h_board, nx, ny);

    return EXIT_SUCCESS;
}


/*************************************************************************************
 * Utility functions
 ************************************************************************************/

// Function to load the params file and set up the X and Y dimensions
void load_params(const char* file, unsigned int *nx, unsigned int *ny, unsigned int *iterations)
{
    FILE *fp = fopen(file, "r");
    if (!fp)
        die("Could not open params file.", __LINE__, __FILE__);

    int retval;
    retval = fscanf(fp, "%d\n", nx);
    if (retval != 1)
        die("Could not read params file: nx.", __LINE__, __FILE__);
    retval = fscanf(fp, "%d\n", ny);
    if (retval != 1)
        die("Could not read params file: ny", __LINE__, __FILE__);
    retval = fscanf(fp, "%d\n", iterations);
    if (retval != 1)
        die("Could not read params file: iterations", __LINE__, __FILE__);

    fclose(fp);
}

// Function to load in a file which lists the alive cells
// Each line of the file is expected to be: x y 1
void load_board(char* board, const char* file, const unsigned int nx, const unsigned int ny)
{
    FILE *fp = fopen(file, "r");
    if (!fp)
        die("Could not open input file.", __LINE__, __FILE__);

    int retval;
    unsigned int x, y, s;
    while ((retval = fscanf(fp, "%d %d %d\n", &x, &y, &s)) != EOF)
    {
        if (retval != 3)
            die("Expected 3 values per line in input file.", __LINE__, __FILE__);
        if (x < 0 || x > nx - 1)
            die("Input x-coord out of range.", __LINE__, __FILE__);
        if (y < 0 || y > ny - 1)
            die("Input y-coord out of range.", __LINE__, __FILE__);
        if (s != ALIVE)
            die("Alive value should be 1.", __LINE__, __FILE__);

        board[x + y * nx] = ALIVE;
    }

    fclose(fp);
}

// Function to print out the board to stdout
// Alive cells are displayed as O
// Dead cells are displayed as .
void print_board(const char* board, const unsigned int nx, const unsigned int ny)
{
    for (unsigned int i = 0; i < ny; i++)
    {
        for (unsigned int j = 0; j < nx; j++)
        {
            if (board[i * nx + j] == DEAD)
                printf(".");
            else
                printf("O");
        }
        printf("\n");
    }
}

void save_board(const char* board, const unsigned int nx, const unsigned int ny)
{
    FILE *fp = fopen(FINALSTATEFILE, "w");
    if (!fp)
        die("Could not open final state file.", __LINE__, __FILE__);

    for (unsigned int i = 0; i < ny; i++)
    {
        for (unsigned int j = 0; j < nx; j++)
        {
            if (board[i * nx + j] == ALIVE)
                fprintf(fp, "%d %d %d\n", j, i, ALIVE);
        }
    }
}

void errorCheck(hipError_t error)
{
    if (error != hipSuccess)
        die(hipGetErrorString(error), __LINE__, __FILE__);
}

// Function to display error and exit nicely
void die(const char* message, const int line, const char *file)
{
  fprintf(stderr, "Error at line %d of file %s:\n", line, file);
  fprintf(stderr, "%s\n",message);
  fflush(stderr);
  exit(EXIT_FAILURE);
}